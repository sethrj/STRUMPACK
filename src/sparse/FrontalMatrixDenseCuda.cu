#include "hip/hip_runtime.h"
/*
 * STRUMPACK -- STRUctured Matrices PACKage, Copyright (c) 2014, The
 * Regents of the University of California, through Lawrence Berkeley
 * National Laboratory (subject to receipt of any required approvals
 * from the U.S. Dept. of Energy).  All rights reserved.
 *
 * If you have questions about your rights to use or distribute this
 * software, please contact Berkeley Lab's Technology Transfer
 * Department at TTD@lbl.gov.
 *
 * NOTICE. This software is owned by the U.S. Department of Energy. As
 * such, the U.S. Government has been granted for itself and others
 * acting on its behalf a paid-up, nonexclusive, irrevocable,
 * worldwide license in the Software to reproduce, prepare derivative
 * works, and perform publicly and display publicly.  Beginning five
 * (5) years after the date permission to assert copyright is obtained
 * from the U.S. Department of Energy, and subject to any subsequent
 * five (5) year renewals, the U.S. Government is granted for itself
 * and others acting on its behalf a paid-up, nonexclusive,
 * irrevocable, worldwide license in the Software to reproduce,
 * prepare derivative works, distribute copies to the public, perform
 * publicly and display publicly, and to permit others to do so.
 *
 * Developers: Pieter Ghysels, Francois-Henry Rouet, Xiaoye S. Li.
 *             (Lawrence Berkeley National Lab, Computational Research
 *             Division).
 *
 */
#ifndef CUDAWRAPPER_H
#define CUDAWRAPPER_H

#include <cmath>
#include <complex>
#include <iostream>
#include <cassert>
#include <memory>
//#include "../dense/BLASLAPACKWrapper.hpp"
// #include "StrumpackParameters.hpp"
// #include "StrumpackFortranCInterface.h"
#include "FrontalMatrixDenseCudaWrapper.h"
//#include <hipsolver.h>
//#include "hipblas.h"
//#include <hip/hip_runtime.h>
#define TILE_DIM 8
//#endif

namespace strumpack {

  namespace cuda {
    ///////////////////////////////////////////////////////////
    ///////// CUBLAS //////////////////////////////////////////
    ///////////////////////////////////////////////////////////
    __host__ inline hipblasStatus_t cublasgemmwrapper
    (hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
     int m, int n, int k, const float *alpha, const float *A, int lda,
     const float *B, int ldb, const float *beta, float *C, int ldc) {
      hipblasStatus_t stat = hipblasSgemm(handle, transa, transb, m, n, k, alpha, A, lda, B,ldb, beta, 
        C, ldc);
      return stat;
    }
    __host__ inline hipblasStatus_t cublasgemmwrapper
    (hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
     int m, int n, int k, const double *alpha, const double *A, int lda,
     const double *B, int ldb, const double *beta, double *C, int ldc) {
      hipblasStatus_t stat = hipblasDgemm(handle, transa, transb, m, n, k, alpha, A, lda,  B, ldb, beta, C, ldc);
      return stat;
    }
    __host__ inline hipblasStatus_t cublasgemmwrapper
    (hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
     int m, int n, int k, const std::complex<float> *alpha, const std::complex<float> *A, 
     int lda, const std::complex<float> *B, int ldb, const std::complex<float> *beta, 
     std::complex<float> *C, int ldc) {
      const hipComplex* c_alpha = reinterpret_cast<const hipComplex*>(alpha);
      const hipComplex* c_A = reinterpret_cast<const hipComplex*>(A);
      const hipComplex* c_B = reinterpret_cast<const hipComplex*>(B);
      const hipComplex* c_beta = reinterpret_cast<const hipComplex*>(beta);
      hipComplex* c_C = reinterpret_cast<hipComplex*>(C);
    
      hipblasStatus_t stat = hipblasCgemm(handle, transa, transb, m, n, k, c_alpha, c_A, lda, c_B, ldb, c_beta, c_C, ldc);
      return stat;
    }
    __host__ inline hipblasStatus_t cublasgemmwrapper
    (hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
     int m, int n, int k, const std::complex<double> *alpha, const std::complex<double> *A, 
     int lda, const std::complex<double> *B, int ldb, const std::complex<double> *beta, 
     std::complex<double> *C, int ldc) {
      const hipDoubleComplex* c_alpha = reinterpret_cast<const hipDoubleComplex*>(alpha);
      const hipDoubleComplex* c_A = reinterpret_cast<const hipDoubleComplex*>(A);
      const hipDoubleComplex* c_B = reinterpret_cast<const hipDoubleComplex*>(B);
      const hipDoubleComplex* c_beta = reinterpret_cast<const hipDoubleComplex*>(beta);
      hipDoubleComplex* c_C = reinterpret_cast<hipDoubleComplex*>(C);
    
      hipblasStatus_t stat = hipblasZgemm(handle, transa, transb, m, n, k, c_alpha, c_A, lda, c_B, ldb, c_beta, 
        c_C, ldc);
      return stat;
    }
    hipblasStatus_t cublasgemm
    (hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
     int m, int n, int k, const float *alpha, const float *A, int lda,
     const float *B, int ldb, const float *beta, float *C, int ldc) {
      hipblasStatus_t stat = cublasgemmwrapper(handle, transa, transb, m, n, k, alpha, A, lda, 
                                              B, ldb, beta, C, ldc);
      return stat;
    }
    hipblasStatus_t cublasgemm
    (hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
     int m, int n, int k, const double *alpha, const double *A, int lda,
     const double *B, int ldb, const double *beta, double *C, int ldc) {
       hipblasStatus_t stat = cublasgemmwrapper(handle, transa, transb, m, n, k, alpha, A, lda, 
                                               B, ldb, beta, C, ldc);
       return stat;
    }
    hipblasStatus_t cublasgemm
    (hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
     int m, int n, int k, const std::complex<float> *alpha, const std::complex<float> *A, 
     int lda, const std::complex<float> *B, int ldb, const std::complex<float> *beta, 
     std::complex<float> *C, int ldc) {
       hipblasStatus_t stat = cublasgemmwrapper(handle, transa, transb, m, n, k, alpha, A, lda, 
                                               B, ldb, beta, C, ldc);
       return stat;
    }
    hipblasStatus_t cublasgemm
    (hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
     int m, int n, int k, const std::complex<double> *alpha, const std::complex<double> *A, 
     int lda, const std::complex<double> *B, int ldb, const std::complex<double> *beta, 
     std::complex<double> *C, int ldc) {
       hipblasStatus_t stat = cublasgemmwrapper(handle, transa, transb, m, n, k, alpha, A, lda, 
                                               B, ldb, beta, C, ldc);
       return stat;
    }

    ///////////////////////////////////////////////////////////
    ///////// CUSOLVER ////////////////////////////////////////
    ///////////////////////////////////////////////////////////
    __host__ inline hipsolverStatus_t cusolverDngetrf_buffersizewrapper
    (hipsolverHandle_t handle, int m, int n, float *A, int lda, int *Lwork) {
      hipsolverStatus_t stat = hipsolverDnSgetrf_bufferSize(handle, m, n, A, lda, Lwork);
      return stat;
    }
    __host__ inline hipsolverStatus_t cusolverDngetrf_buffersizewrapper
    (hipsolverHandle_t handle, int m, int n, double *A, int lda, int *Lwork) {
      hipsolverStatus_t stat = hipsolverDnDgetrf_bufferSize(handle, m, n, A, lda, Lwork);
      return stat;
    }
    __host__ inline hipsolverStatus_t cusolverDngetrf_buffersizewrapper
    (hipsolverHandle_t handle, int m, int n, std::complex<float> *A, int lda, int *Lwork) {
      hipComplex* c_A = reinterpret_cast<hipComplex*>(A);
      hipsolverStatus_t stat = hipsolverDnCgetrf_bufferSize(handle, m, n, c_A, lda, Lwork);
      return stat;
    }
    __host__ inline hipsolverStatus_t cusolverDngetrf_buffersizewrapper
    (hipsolverHandle_t handle, int m, int n, std::complex<double> *A, int lda, int *Lwork) {
      hipDoubleComplex* c_A = reinterpret_cast<hipDoubleComplex*>(A);
      hipsolverStatus_t stat = hipsolverDnZgetrf_bufferSize(handle, m, n, c_A, lda, Lwork);
      return stat;
    }

    hipsolverStatus_t cusolverDngetrf_bufferSize
    (hipsolverHandle_t handle, int m, int n, float *A, int lda, int *Lwork) {
      hipsolverStatus_t stat = cusolverDngetrf_buffersizewrapper(handle, m, n, A, lda, Lwork);
      return stat;
    }
    hipsolverStatus_t cusolverDngetrf_bufferSize
    (hipsolverHandle_t handle, int m, int n, double *A, int lda, int *Lwork) {
      hipsolverStatus_t stat = cusolverDngetrf_buffersizewrapper(handle, m, n, A, lda, Lwork);
      return stat;
    }
    hipsolverStatus_t cusolverDngetrf_bufferSize
    (hipsolverHandle_t handle, int m, int n, std::complex<float> *A, int lda, int *Lwork) {
      hipsolverStatus_t stat = cusolverDngetrf_buffersizewrapper(handle, m, n, A, lda, Lwork);
      return stat;
    }
    hipsolverStatus_t cusolverDngetrf_bufferSize
    (hipsolverHandle_t handle, int m, int n, std::complex<double> *A, int lda, int *Lwork) {
      hipsolverStatus_t stat = cusolverDngetrf_buffersizewrapper(handle, m, n, A, lda, Lwork);
      return stat;
    }

    __host__ inline hipsolverStatus_t cusolverDngetrfwrapper
    (hipsolverHandle_t handle, int m, int n, float *A, int lda, float *Workspace, 
     int *devIpiv, int *devInfo) {
       hipsolverStatus_t stat = hipsolverDnSgetrf(handle, m, n, A, lda, Workspace, devIpiv, devInfo);
       return stat;
    }
    __host__ inline hipsolverStatus_t cusolverDngetrfwrapper
    (hipsolverHandle_t handle, int m, int n, double *A, int lda, double *Workspace, 
     int *devIpiv, int *devInfo) {
       hipsolverStatus_t stat = hipsolverDnDgetrf(handle, m, n, A, lda, Workspace, devIpiv, devInfo);
       return stat;
    }
    __host__ inline hipsolverStatus_t cusolverDngetrfwrapper
    (hipsolverHandle_t handle, int m, int n, std::complex<float> *A, int lda, std::complex<float> *Workspace, 
     int *devIpiv, int *devInfo) {
       hipComplex* c_A = reinterpret_cast<hipComplex*>(A);
       hipComplex* c_Workspace = reinterpret_cast<hipComplex*>(Workspace);
       hipsolverStatus_t stat = hipsolverDnCgetrf(handle, m, n, c_A, lda, c_Workspace, devIpiv, devInfo);
       return stat;
    }
    __host__ inline hipsolverStatus_t cusolverDngetrfwrapper
    (hipsolverHandle_t handle, int m, int n, std::complex<double> *A, int lda, std::complex<double> *Workspace, 
     int *devIpiv, int *devInfo) {
       hipDoubleComplex* c_A = reinterpret_cast<hipDoubleComplex*>(A);
       hipDoubleComplex* c_Workspace = reinterpret_cast<hipDoubleComplex*>(Workspace);
       hipsolverStatus_t stat = hipsolverDnZgetrf(handle, m, n, c_A, lda, c_Workspace, devIpiv, devInfo);
       return stat;
    }

    hipsolverStatus_t cusolverDngetrf
    (hipsolverHandle_t handle, int m, int n, float *A, int lda, float *Workspace, 
     int *devIpiv, int *devInfo) {
       hipsolverStatus_t stat = cusolverDngetrfwrapper(handle, m, n, A, lda, Workspace, 
                                                      devIpiv, devInfo);
       return stat;
    }
    hipsolverStatus_t cusolverDngetrf
    (hipsolverHandle_t handle, int m, int n, double *A, int lda, double *Workspace, 
     int *devIpiv, int *devInfo) {
       hipsolverStatus_t stat = cusolverDngetrfwrapper(handle, m, n, A, lda, Workspace, 
                                                      devIpiv, devInfo);
       return stat;
    }
    hipsolverStatus_t cusolverDngetrf
    (hipsolverHandle_t handle, int m, int n, std::complex<float> *A, int lda, std::complex<float> *Workspace, 
     int *devIpiv, int *devInfo) {
       hipsolverStatus_t stat = cusolverDngetrfwrapper(handle, m, n, A, lda, Workspace, 
                                                      devIpiv, devInfo);
       return stat;
    }
    hipsolverStatus_t cusolverDngetrf
    (hipsolverHandle_t handle, int m, int n, std::complex<double> *A, int lda, std::complex<double> *Workspace, 
     int *devIpiv, int *devInfo) {
       hipsolverStatus_t stat = cusolverDngetrfwrapper(handle, m, n, A, lda, Workspace, 
                                                      devIpiv, devInfo);
       return stat;
    }

    __host__ inline hipsolverStatus_t cusolverDngetrswrapper
    (hipsolverHandle_t handle, hipblasOperation_t trans, int n, int nrhs, const float *A, 
     int lda, const int *devIpiv, float *B, int ldb, int *devInfo) {
       hipsolverStatus_t stat = hipsolverDnSgetrs(handle, trans, n, nrhs, A, lda, devIpiv, B, ldb, devInfo);
       return stat;
    }
    __host__ inline hipsolverStatus_t cusolverDngetrswrapper
    (hipsolverHandle_t handle, hipblasOperation_t trans, int n, int nrhs, const double *A, 
     int lda, const int *devIpiv, double *B, int ldb, int *devInfo) {
       hipsolverStatus_t stat = hipsolverDnDgetrs(handle, trans, n, nrhs, A, lda, devIpiv, B, ldb, devInfo);
       return stat;
    }
    __host__ inline hipsolverStatus_t cusolverDngetrswrapper
    (hipsolverHandle_t handle, hipblasOperation_t trans, int n, int nrhs, const std::complex<float> *A, 
     int lda, const int *devIpiv, std::complex<float> *B, int ldb, int *devInfo) {
       const hipComplex* c_A = reinterpret_cast<const hipComplex*>(A); 
       hipComplex* c_B = reinterpret_cast<hipComplex*>(B); 
       hipsolverStatus_t stat = hipsolverDnCgetrs(handle, trans, n, nrhs, c_A, lda, devIpiv, c_B, ldb, devInfo);
       return stat;
    }
    __host__ inline hipsolverStatus_t cusolverDngetrswrapper
    (hipsolverHandle_t handle, hipblasOperation_t trans, int n, int nrhs, const std::complex<double> *A, 
     int lda, const int *devIpiv, std::complex<double> *B, int ldb, int *devInfo) {
       const hipDoubleComplex* c_A = reinterpret_cast<const hipDoubleComplex*>(A); 
       hipDoubleComplex* c_B = reinterpret_cast<hipDoubleComplex*>(B); 
       hipsolverStatus_t stat = hipsolverDnZgetrs(handle, trans, n, nrhs, c_A, lda, devIpiv, c_B, ldb, devInfo);
       return stat;
    }

    hipsolverStatus_t cusolverDngetrs
    (hipsolverHandle_t handle, hipblasOperation_t trans, int n, int nrhs, const float *A, 
     int lda, const int *devIpiv, float *B, int ldb, int *devInfo) {
      hipsolverStatus_t stat = cusolverDngetrswrapper
        (handle, trans, n, nrhs, A, lda, devIpiv, B, ldb, devInfo);
      return stat;
    }
    hipsolverStatus_t cusolverDngetrs
    (hipsolverHandle_t handle, hipblasOperation_t trans, int n, int nrhs, const double *A, 
     int lda, const int *devIpiv, double *B, int ldb, int *devInfo) {
      hipsolverStatus_t stat = cusolverDngetrswrapper
        (handle, trans, n, nrhs, A, lda, devIpiv, B, ldb, devInfo);
      return stat;
    }
    hipsolverStatus_t cusolverDngetrs
    (hipsolverHandle_t handle, hipblasOperation_t trans, int n, int nrhs, const std::complex<float> *A, 
     int lda, const int *devIpiv, std::complex<float> *B, int ldb, int *devInfo) {
      hipsolverStatus_t stat = cusolverDngetrswrapper
        (handle, trans, n, nrhs, A, lda, devIpiv, B, ldb, devInfo);
      return stat;
    }
    hipsolverStatus_t cusolverDngetrs
    (hipsolverHandle_t handle, hipblasOperation_t trans, int n, int nrhs, const std::complex<double> *A, 
     int lda, const int *devIpiv, std::complex<double> *B, int ldb, int *devInfo) {
      hipsolverStatus_t stat = cusolverDngetrswrapper
        (handle, trans, n, nrhs, A, lda, devIpiv, B, ldb, devInfo);
      return stat;
    }
    ///////////////////////////////////////////////////////////
    ///////// KERNELS /////////////////////////////////////////
    ///////////////////////////////////////////////////////////

    __device__ void swap(size_t& a, size_t& b) {
      size_t tmp = a;
      a = b;
      b = tmp;
    }
    __device__ void swap(int& a, int& b) {
      int tmp = a;
      a = b;
      b = tmp;
    }
    __device__ void swap(double& a, double& b) {
      double& tmp = a;
      a = b;
      b = tmp;
    }

    __global__ void partialLU
    (int* l_n1, int* l_n2, double** l_A11, double** l_A12, double** l_A21, double** l_A22, int** l_piv) {

      int t_id_x = threadIdx.x;
      int t_id_y = threadIdx.y;
      int blkdim_x = blockDim.x;
      int blkdim_y = blockDim.y;

      int n1 = l_n1[blockIdx.x];
      int n2 = l_n2[blockIdx.x];
      double* A11 = l_A11[blockIdx.x];
      double* A12 = l_A12[blockIdx.x];
      double* A21 = l_A21[blockIdx.x];
      double* A22 = l_A22[blockIdx.x];
      int* piv = l_piv[blockIdx.x]; 

      if (t_id_y == 0)
        for (int i=t_id_x; i<n1; i+=blkdim_x)
          piv[i] = i+1; // fortran convention
      for (int j=0; j<n1; j++) {
        auto Amax = A11[j+j*n1];
        int imax = j;
        if (t_id_x == 0 && t_id_y == 0) {
          // find pivot element
          for (int i=j+1; i<n1; i++) {
            if (fabs(A11[i+j*n1]) > fabs(Amax)) {
              Amax = A11[i+j*n1];
              imax = i;
            }
          }
          //if (Amax == 0) return j;
          if (imax != j)
            cuda::swap(piv[j], piv[imax]);
        }
	__syncthreads();
        if (imax != j) {
          if (t_id_y == 0)
            for (int i=t_id_x; i<n1; i+=blkdim_x)
              cuda::swap(A11[imax+i*n1], A11[j+i*n1]);
          if (t_id_y == 1)
            for (int i=t_id_x; i<n2; i+=blkdim_x)
              cuda::swap(A12[imax+i*n1], A12[j+i*n1]);
        }
        __syncthreads();
        auto iAmax = 1.0 / Amax;
        if (t_id_y == 0)
          for (int i=j+1+t_id_x; i<n1; i+=blkdim_x)
	    A11[i+j*n1] *= iAmax;
        __syncthreads();
        for (int i=j+1+t_id_x; i<n1; i+=blkdim_x) {
          for (int k=j+1+t_id_y; k<n1; k+=blkdim_y)
            A11[k+i*n1] -= A11[k+j*n1] * A11[j+i*n1];
        }
      }
      __syncthreads();
      // trsm with L and U
      for (int k=t_id_x + blkdim_x*t_id_y; k<n2; k+=blkdim_x*blkdim_y) {
        for (int i=0; i<n1; i++)
          for (int j=0; j<i; j++)
            A12[i+k*n1] -= A11[i+j*n1] * A12[j+k*n1];
        for (int i=n1-1; i>=0; i--) {
          for (int j=i+1; j<n1; j++)
            A12[i+k*n1] -= A11[i+j*n1] * A12[j+k*n1];
          A12[i+k*n1] /= A11[i+i*n1];
        }
      }
      __syncthreads();
      // gemm
      for (int j=t_id_x; j<n2; j+=blkdim_x)
        for (int i=t_id_y; i<n2; i+=blkdim_y)
          for (int k=0; k<n1; k++)
            A22[i+j*n2] -= A21[i+k*n2] * A12[k+j*n1];
    }

    void partialLUWrapper
    (int num_blocks, dim3 threads_per_block, int* l_n1, 
     int* l_n2, double** l_A11, double** l_A12, double** l_A21, double** l_A22, int** l_piv) {
       partialLU<<<num_blocks,threads_per_block>>>
         (l_n1, l_n2, l_A11, l_A12, l_A21, l_A22, l_piv);
    }
  }


    __global__ void LUkernel
    (int* l_n1, int* l_n2, double** l_A11, double** l_A12, double** l_A21, double** l_A22, int** l_piv) {

      int t_id_x = threadIdx.x;
      int t_id_y = threadIdx.y;
      int blkdim_x = blockDim.x;
      int blkdim_y = blockDim.y;

      int n1 = l_n1[blockIdx.x];
      int n2 = l_n2[blockIdx.x];
      double* A11 = l_A11[blockIdx.x];
      double* A12 = l_A12[blockIdx.x];
      double* A21 = l_A21[blockIdx.x];
      double* A22 = l_A22[blockIdx.x];
      int* piv = l_piv[blockIdx.x]; 

      if (t_id_y == 0)
        for (int i=t_id_x; i<n1; i+=blkdim_x)
          piv[i] = i+1; // fortran convention
      for (int j=0; j<n1; j++) {
        auto Amax = A11[j+j*n1];
        int imax = j;
        if (t_id_x == 0 && t_id_y == 0) {
          // find pivot element
          for (int i=j+1; i<n1; i++) {
            if (fabs(A11[i+j*n1]) > fabs(Amax)) {
              Amax = A11[i+j*n1];
              imax = i;
            }
          }
          //if (Amax == 0) return j;
          if (imax != j)
            cuda::swap(piv[j], piv[imax]);
        }
	__syncthreads();
        if (imax != j) {
          if (t_id_y == 0)
            for (int i=t_id_x; i<n1; i+=blkdim_x)
              cuda::swap(A11[imax+i*n1], A11[j+i*n1]);
          if (t_id_y == 1)
            for (int i=t_id_x; i<n2; i+=blkdim_x)
              cuda::swap(A12[imax+i*n1], A12[j+i*n1]);
        }
        __syncthreads();
        auto iAmax = 1.0 / Amax;
        if (t_id_y == 0)
          for (int i=j+1+t_id_x; i<n1; i+=blkdim_x)
	    A11[i+j*n1] *= iAmax;
        __syncthreads();
        for (int i=j+1+t_id_x; i<n1; i+=blkdim_x) {
          for (int k=j+1+t_id_y; k<n1; k+=blkdim_y)
            A11[k+i*n1] -= A11[k+j*n1] * A11[j+i*n1];
        }
      }
    }

    void LUkernelWrapper
    (int num_blocks, dim3 threads_per_block, int* l_n1, 
     int* l_n2, double** l_A11, double** l_A12, double** l_A21, double** l_A22, int** l_piv) {
       LUkernel<<<num_blocks,threads_per_block>>>
         (l_n1, l_n2, l_A11, l_A12, l_A21, l_A22, l_piv);
    }
  }

}
#endif
